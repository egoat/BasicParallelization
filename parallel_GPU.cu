#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <omp.h>
//#include <mpi.h>
#include <time.h>

#include "config.c"

// function to add the elements of two arrays
// CUDA kernel function to add the elements of two arrays on the GPU
__global__
void add(int n, float *x, float *y)
{
    int i=0;
    //#pragma omp parallel for private(i) num_threads(4)
    for (i = 0; i < n; i++)
    {
        y[i] = x[i] + y[i];
    }
}

int main()
{

    //
    //
    //
    
    hipMallocManaged(&x,N*sizeof(*x));
    hipMallocManaged(&y,N*sizeof(*y));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0;
        y[i] = 2.0;
    }

    //=========================================================
    // Run kernel on 1M elements on the CPU
    //-------------------------------------
//    start = timeInMilliseconds();
    //----------
    //for (int i = 0; i<rep; i++)
    //{
        //add(N, x, y);
        add<<<1, 1>>>(N,x,y);
        hipDeviceSynchronize();
    //}
    //-----------
//    end = timeInMilliseconds();
    //-------------------------------------
    //=========================================================

//    cpu_time_used = ((double) (end-start));
    cpu_time_used = 0;

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0;
    for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-(2.0+rep)));
    printf("1\t%f\t%f\n",cpu_time_used,maxError);

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;

}