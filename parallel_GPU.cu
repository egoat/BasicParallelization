#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <omp.h>
//#include <mpi.h>
#include <time.h>

#include "config.c"

// function to add the elements of two arrays
// CUDA kernel function to add the elements of two arrays on the GPU
__global__
void add(int n, float *x, float *y)
{
    int t = threadIdx.x;
    int T = blockDim.x;
    for (int i = t; i < n; i += T)
    {
        y[i] = x[i] + y[i];
    }
}

int main()
{
    
    hipMallocManaged(&x,N*sizeof(*x));
    hipMallocManaged(&y,N*sizeof(*y));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0;
        y[i] = 2.0;
    }

    //=========================================================
    // Run kernel on 1M elements on the CPU
    //-------------------------------------
    start = timeInMilliseconds();
    //----------
    for (int i = 0; i<rep; i++)
    {
        add<<<1, 512>>>(N,x,y);
        hipDeviceSynchronize();
    }
    //-----------
    end = timeInMilliseconds();
    //-------------------------------------
    //=========================================================

    cpu_time_used = ((double) (end-start));
//    cpu_time_used = 0;

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0;
    for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-(2.0+rep)));
    printf("1\t%f\t%f\n",cpu_time_used,maxError);

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;

}